#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// CUDA kernel to calculate the sum of a series of numbers
__global__ void sumOfSeries(float* result, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0.0f;

    while (tid < n) {
        sum += tid;
        tid += blockDim.x * gridDim.x;
    }

    atomicAdd(result, sum);
}

int main() {
    int n;

    // Prompt the user to enter the number of elements in the series
    std::cout << "Enter the number of elements in the series: ";
    std::cin >> n;

    int numThreadsPerBlock = 256;
    int numBlocks = (n + numThreadsPerBlock - 1) / numThreadsPerBlock;

    // Allocate memory on the CPU to store the result
    float* h_result = new float;
    *h_result = 0.0f;

    // Allocate memory on the GPU to store the result
    float* d_result;
    hipMalloc((void**)&d_result, sizeof(float));
    hipMemcpy(d_result, h_result, sizeof(float), hipMemcpyHostToDevice);

    // Start time measurement
    auto start = std::chrono::high_resolution_clock::now();

    // Launch the CUDA kernel
    sumOfSeries<<<numBlocks, numThreadsPerBlock>>>(d_result, n);

    // Ensure that the GPU has completed its work
    hipDeviceSynchronize();

    // End time measurement
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    std::cout << "Time taken for execution: " << duration.count() << " nanoseconds" << std::endl;

    // Copy the result back to the CPU
    hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Print the sum of the series
    std::cout << "Sum of the series: " << *h_result << std::endl;

    // Cleanup
    delete h_result;
    hipFree(d_result);

    return 0;
}

